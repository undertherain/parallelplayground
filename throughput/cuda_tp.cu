#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <unistd.h>
#include <sys/times.h>
#include <omp.h>

typedef unsigned long long Index;

int main()
{
  	int devCount;
    hipGetDeviceCount(&devCount); //random thing to avoid latency of a first call to device

	size_t size=2LL*1024*1024*1024;
	unsigned char *cpuBuf = new unsigned char[size];
	unsigned char * gpuBuf;
    clock_t clockStart, clockStop;
    tms tmsStart, tmsStop;

    std::cerr<<"creating array of random numbers \n";
    std::cerr<<"size = "<<static_cast<double>(size)/(1024*1024*1024)<<"G \n";

 	clockStart = times(&tmsStart);
    unsigned char * array = new unsigned char [size];
    for (Index i=0;i<size;i++)
        array[i]= rand()%256; //not the best way to generate random nubers
    clockStop = times(&tmsStop);
    std::cerr << "Done in " << (clockStop - clockStart)/static_cast<double>(sysconf(_SC_CLK_TCK)) << " seconds\n\n";
    std::cerr<<"offloading to GPU \n";


    clockStart = times(&tmsStart);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMalloc((void**)&gpuBuf, size);
	hipEventRecord(start, 0);
    hipMemcpy(gpuBuf,cpuBuf, size,hipMemcpyHostToDevice);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

    clockStop = times(&tmsStop);
    std::cerr << "Done in " ;
    double secs= (clockStop - clockStart)/static_cast<double>(sysconf(_SC_CLK_TCK));
    std::cerr << secs << " seconds\n" << std::endl;

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
	std::cout<<"elapsed gpu time= "<<elapsedTime/1000<<"s"<<std::endl;

	hipFree (gpuBuf);
	delete[] cpuBuf;
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return 0;
}
